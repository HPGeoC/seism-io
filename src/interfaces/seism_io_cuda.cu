/**
@section LICENSE
Copyright (c) 2013-2017, Regents of the University of California
All rights reserved.
Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/*************************************************************************
	> File Name: seism_io_cuda.cu
	> Author: huizhou
	> Mail: wbzhui@gmail.com
	> Created Time: Wed 07 Sep 2016 10:47:41 AM PDT
 ************************************************************************/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<assert.h>
#include "seism_io_globals.h"
#include "seism_io_consts.h"

int bufferfromGPU = 0;

/*
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}
*/

void *convertCudaPointer(FileProfile *fp, void *buffer, int *err)
{

    hipError_t code;

    //hipDeviceProp_t prop;
    //code = hipGetDeviceProperties(&prop , 0);

    hipPointerAttribute_t attributes;
    code =hipPointerGetAttributes (&attributes, buffer);
    if (code != hipSuccess)
    {
        fprintf(stderr,"hipPointerGetAttributes error: %s.\n", hipGetErrorString(code));
        *err = -1;
        return NULL;
    }

    //if buffer comes from GPU Space
    if(attributes.memoryType==hipMemoryTypeDevice)
    {
        bufferfromGPU = 1;
        PointSet *ps = fp->pointSet;
        if(!ps)
        {
            printf("ERROR! SEISM_IO: no PS in FP!\n");
            *err = -2;
            return NULL;
        }

    	//void* d_data = (void*)hipHostMalloc(sizeof(fp->datatype)*(ps->nprec)*(fp->write_step));
        void* m_data = (void*)malloc(sizeof(fp->datatype)*(ps->nprec)*(fp->write_step));
        //if (code != hipSuccess)
        if (m_data != NULL)
        {
            //fprintf(stderr,"hipHostMalloc error: %s.\n", hipGetErrorString(code));
            fprintf(stderr,"Malloc error.\n");
            *err = -3;
            return NULL;
        }

        code = hipMemcpy(m_data, buffer, sizeof(fp->datatype)*(ps->nprec)*(fp->write_step), hipMemcpyDeviceToHost);
        if (code != hipSuccess)
        {
            fprintf(stderr,"hipMemcpy error: %s.\n", hipGetErrorString(code));
            //hipFree(buffer);
            free(buffer);
            buffer = NULL;
            *err = -4;
            return NULL;
        }
        *err = 0;
        return m_data;

    }
    //buffer comes from Host
    else
    {
        bufferfromGPU = 0;
        return buffer;
    }

}

void freeCudaPointer(void *buffer)
{
    if(bufferfromGPU = 1 && buffer != NULL)
    {
        //hipFree(buffer);
        free(buffer);
        buffer = NULL;
    }
}
